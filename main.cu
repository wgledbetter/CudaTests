
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mykernel(void) {

}

__global__ void add(int *a, int *b, int *c){
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

#define N 512

int main(void){
    mykernel<<<1,1>>>();
    std::cout << "wow" << std::endl;

    int *a, *b, *c;  // Host variables
    int *x, *y, *z;  // Device variables
    int size = N * sizeof(int);

    hipMalloc((void **)&x, size);
    hipMalloc((void **)&y, size);
    hipMalloc((void **)&z, size);

    // Set Input
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);
    a[23] = 33;
    b[23] = -30;

    // Copy from host to device
    hipMemcpy(x, a, size, hipMemcpyHostToDevice);
    hipMemcpy(y, b, size, hipMemcpyHostToDevice);

    // Run the 'add' kernel
    add<<<1,N>>>(x, y, z);

    // Copy answer from device to host
    hipMemcpy(c, z, size, hipMemcpyDeviceToHost);

    // Clean device
    hipFree(x);
    hipFree(y);
    hipFree(z);

    // Clean Host
    free(a);
    free(b);
    free(c);


    std::cout << "Answer: " << c[23] << std::endl;

    return 12;
}