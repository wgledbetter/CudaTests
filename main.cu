
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mykernel(void) {

}

__global__ void add(int *a, int *b, int *c){
    *c = *a + *b;
}

int main(void){
    mykernel<<<1,1>>>();
    std::cout << "wow" << std::endl;

    int a, b, c;  // Host variables
    int *x, *y, *z;  // Device variables
    int size = sizeof(int);

    hipMalloc((void **)&x, size);
    hipMalloc((void **)&y, size);
    hipMalloc((void **)&z, size);

    a = 2;
    b = 7;

    // Copy from host to device
    hipMemcpy(x, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(y, &b, size, hipMemcpyHostToDevice);

    // Run the 'add' kernel
    add<<<1,1>>>(x, y, z);

    // Copy answer from device to host
    hipMemcpy(&c, z, size, hipMemcpyDeviceToHost);

    // Clean device
    hipFree(x);
    hipFree(y);
    hipFree(z);

    std::cout << "Answer: " << c << std::endl;

    return 12;
}