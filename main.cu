#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

// #include "MyClass.h"
#include "MyCudaClass.cu"

#define N 2560
#define M 512
#define BLOCK_SIZE (N/M)
#define RADIUS 5

__host__ __device__ double hostSumFunction(int n, double m){
    return n+m;
}

struct MyStruct {
    __device__ __host__ MyStruct(int a, double b){
        x = a;
        y = b;
    }

    __device__ __host__ ~MyStruct(){

    }

    __device__ __host__ double get_sum(){
        return hostSumFunction(x, y);
    }

    int x;
    double y;
};

__global__ void add(double *a, double *b, double *c, int n){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n){
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void cudaStructs(MyStruct *ms, double *ans){
    *ans = ms->get_sum();
}

int main(void){
    std::cout << "wow" << std::endl;

    double *a, *b, *c;  // Host variables
    double *x, *y, *z;  // Device variables
    const size_t size = size_t(N) * sizeof(double);

    hipMalloc((void **)&x, size);
    hipMalloc((void **)&y, size);
    hipMalloc((void **)&z, size);

    // Set Input
    a = (double *)malloc(size);
    b = (double *)malloc(size);
    c = (double *)malloc(size);
    a[23] = 33.249;
    b[23] = -30.741;

    // Copy from host to device
    hipMemcpy(x, a, size, hipMemcpyHostToDevice);
    hipMemcpy(y, b, size, hipMemcpyHostToDevice);

    // Run the 'add' kernel
    add<<<N/M, M>>>(x, y, z, N);  // Asynchronous: CPU proceeds before this is finished

    // Copy answer from device to host
    hipMemcpy(c, z, size, hipMemcpyDeviceToHost);  // Waits until all previous CUDA calls have completed

    // Clean device
    hipFree(x);
    hipFree(y);
    hipFree(z);

    // Clean Host
    free(a);
    free(b);
    free(c);

    std::cout << "Answer: " << c[23] << std::endl;

    //==============================================================

    MyStruct *devStruct;
    double hostAns;
    double *devAns;
    const size_t structSize = sizeof(MyStruct);
    const size_t ansSize = sizeof(double);

    hipMalloc((void **)&devStruct, structSize);
    hipMalloc((void **)&devAns, ansSize);

    // Set values
    MyStruct hostStruct(9, 3.14);
    std::cout << "Class member function on host: " << hostStruct.get_sum() << std::endl;

    // Copy struct to device
    hipMemcpy(devStruct, &hostStruct, structSize, hipMemcpyHostToDevice);
    
    // Run the kernel
    cudaStructs<<< 1, 1 >>>(devStruct, devAns);

    // Get answer back to host
    hipMemcpy(&hostAns, devAns, ansSize, hipMemcpyDeviceToHost);

    // Clean Device
    hipFree(devStruct);
    hipFree(devAns);

    // Answer
    std::cout << "Class member function on device: " << hostAns << std::endl;

    //===============================================================

    MyClass mc;
    mc.set_param(12.5);
    std::cout << "Calling Host class, Host member: " << mc.do_it_on_host() << std::endl;

    MyCudaClass mcc;
    mcc.set_param(12.5);
    std::cout << "Calling Cuda class, Host member: " << mcc.do_it_on_host() << std::endl;
    std::cout << "Calling Cuda class, Device member: " << mcc.do_it_on_device() << std::endl;


    return 12;
}