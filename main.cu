
#include <hip/hip_runtime.h>
#include <iostream>

#define N 2560
#define M 512
#define BLOCK_SIZE (N/M)
#define RADIUS 5

__global__ void mykernel(void) {

}

__global__ void add(double *a, double *b, double *c, int n){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n){
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void stencil_1D(int *in, int *out){
    // This is similar to a convolution in the neural network sense

    __shared__ int temp[BLOCK_SIZE + 2*RADIUS];  // Memory that is shared across threads within a block
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;  // Global index (in)
    int lindex = threadIdx.x + RADIUS;  // Local index (temp)

    // Read inputs into shared memory
    temp[lindex] = in[gindex];
    if(threadIdx.x < RADIUS){
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    // Synchronize (ensure all threads have run above code before any proceed to following)
        // Necessary for filling temp
    __syncthreads();

    // Apply stencil
    int result = 0;
    for(int offset = -RADIUS; offset <= RADIUS; offset++){
        result += temp[lindex + offset];
    }

    // Store
    out[gindex] = result;
}

int main(void){
    mykernel<<<1,1>>>();
    std::cout << "wow" << std::endl;

    double *a, *b, *c;  // Host variables
    double *x, *y, *z;  // Device variables
    int size = N * sizeof(double);

    hipMalloc((void **)&x, size);
    hipMalloc((void **)&y, size);
    hipMalloc((void **)&z, size);

    // Set Input
    a = (double *)malloc(size);
    b = (double *)malloc(size);
    c = (double *)malloc(size);
    a[23] = 33.249;
    b[23] = -30.741;

    // Copy from host to device
    hipMemcpy(x, a, size, hipMemcpyHostToDevice);
    hipMemcpy(y, b, size, hipMemcpyHostToDevice);

    // Run the 'add' kernel
    add<<<N/M, M>>>(x, y, z, N);  // Asynchronous: CPU proceeds before this is finished

    // Copy answer from device to host
    hipMemcpy(c, z, size, hipMemcpyDeviceToHost);  // Waits until all previous CUDA calls have completed

    // Clean device
    hipFree(x);
    hipFree(y);
    hipFree(z);

    // Clean Host
    free(a);
    free(b);
    free(c);


    std::cout << "Answer: " << c[23] << std::endl;

    return 12;
}