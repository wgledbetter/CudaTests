
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#define N 2560
#define M 512
#define BLOCK_SIZE (N/M)
#define RADIUS 5

__host__ __device__ double hostSumFunction(int n, double m){
    return n+m;
}

struct MyStruct {
    __device__ __host__ MyStruct(int a, double b){
        x = a;
        y = b;
    }

    __device__ __host__ ~MyStruct(){

    }

    __device__ __host__ double get_sum(){
        return hostSumFunction(x, y);
    }

    int x;
    double y;
};

class MyClass {
    double hostParam;
    double *devParam;
    const size_t dubSize = sizeof(double);

    int nCudaBlocks = 1;
    int nCudaThreadsPerBlock = 1;

    MyClass(){
        hipMalloc((void **)&devParam, dubSize);
    }

    void set_param(double in){
        hostParam = in;
        hipMemcpy(devParam, &hostParam, dubSize, hipMemcpyHostToDevice);
    }

    double do_it_on_host(){
        double out;
        hostKernel(&hostParam, &out);
        return out;
    }

    double do_it_on_device(){
        double *devOut, out;
        hipMalloc((void **)&devOut, dubSize);
        devKernel<<< nCudaBlocks, nCudaThreadsPerBlock >>>(devParam, devOut);
        hipMemcpy(&out, devOut, dubSize, hipMemcpyDeviceToHost);
        return out;
    }

    __global__ static void devKernel(double *param, double *ans){
        // Cuda implementation
        std::printf("Inside devKernel: ");
        *ans = *param + 3.14;
    }

    void hostKernel(double *param, double *ans){
        // Host implementation
        std::cout << "Inside hostKernel: " << "wow" << std::endl;
        *ans = *param + 3.14;
    }
};

__global__ void add(double *a, double *b, double *c, int n){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n){
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void cudaStructs(MyStruct *ms, double *ans){
    *ans = ms->get_sum();
}

int main(void){
    std::cout << "wow" << std::endl;

    double *a, *b, *c;  // Host variables
    double *x, *y, *z;  // Device variables
    const size_t size = size_t(N) * sizeof(double);

    hipMalloc((void **)&x, size);
    hipMalloc((void **)&y, size);
    hipMalloc((void **)&z, size);

    // Set Input
    a = (double *)malloc(size);
    b = (double *)malloc(size);
    c = (double *)malloc(size);
    a[23] = 33.249;
    b[23] = -30.741;

    // Copy from host to device
    hipMemcpy(x, a, size, hipMemcpyHostToDevice);
    hipMemcpy(y, b, size, hipMemcpyHostToDevice);

    // Run the 'add' kernel
    add<<<N/M, M>>>(x, y, z, N);  // Asynchronous: CPU proceeds before this is finished

    // Copy answer from device to host
    hipMemcpy(c, z, size, hipMemcpyDeviceToHost);  // Waits until all previous CUDA calls have completed

    // Clean device
    hipFree(x);
    hipFree(y);
    hipFree(z);

    // Clean Host
    free(a);
    free(b);
    free(c);

    std::cout << "Answer: " << c[23] << std::endl;

    //==============================================================

    MyStruct *devStruct;
    double hostAns;
    double *devAns;
    const size_t structSize = sizeof(MyStruct);
    const size_t ansSize = sizeof(double);

    hipMalloc((void **)&devStruct, structSize);
    hipMalloc((void **)&devAns, ansSize);

    // Set values
    MyStruct hostStruct(9, 3.14);
    std::cout << "Class member function on host: " << hostStruct.get_sum() << std::endl;

    // Copy struct to device
    hipMemcpy(devStruct, &hostStruct, structSize, hipMemcpyHostToDevice);
    
    // Run the kernel
    cudaStructs<<< 1, 1 >>>(devStruct, devAns);

    // Get answer back to host
    hipMemcpy(&hostAns, devAns, ansSize, hipMemcpyDeviceToHost);

    // Clean Device
    hipFree(devStruct);
    hipFree(devAns);

    // Answer
    std::cout << "Class member function on device: " << hostAns << std::endl;

    //===============================================================



    return 12;
}