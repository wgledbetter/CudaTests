
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mykernel(void) {

}

__global__ void add(int *a, int *b, int *c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 512

int main(void){
    mykernel<<<1,1>>>();
    std::cout << "wow" << std::endl;

    int *a, *b, *c;  // Host variables
    int *x, *y, *z;  // Device variables
    int size = N * sizeof(int);

    hipMalloc((void **)&x, size);
    hipMalloc((void **)&y, size);
    hipMalloc((void **)&z, size);

    // Set Input
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);
    // std::random_ints(a, N);
    // std::random_ints(b, N);

    // Copy from host to device
    hipMemcpy(x, a, size, hipMemcpyHostToDevice);
    hipMemcpy(y, b, size, hipMemcpyHostToDevice);

    // Run the 'add' kernel
    add<<<N,1>>>(x, y, z);

    // Copy answer from device to host
    hipMemcpy(c, z, size, hipMemcpyDeviceToHost);

    // Clean device
    hipFree(x);
    hipFree(y);
    hipFree(z);

    // Clean Host
    free(a);
    free(b);
    free(c);


    std::cout << "Answer: " << c[23] << std::endl;

    return 12;
}