
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mykernel(void) {

}

__global__ void add(double *a, double *b, double *c){
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

#define N 512

int main(void){
    mykernel<<<1,1>>>();
    std::cout << "wow" << std::endl;

    double *a, *b, *c;  // Host variables
    double *x, *y, *z;  // Device variables
    int size = N * sizeof(double);

    hipMalloc((void **)&x, size);
    hipMalloc((void **)&y, size);
    hipMalloc((void **)&z, size);

    // Set Input
    a = (double *)malloc(size);
    b = (double *)malloc(size);
    c = (double *)malloc(size);
    a[23] = 33.249;
    b[23] = -30.741;

    // Copy from host to device
    hipMemcpy(x, a, size, hipMemcpyHostToDevice);
    hipMemcpy(y, b, size, hipMemcpyHostToDevice);

    // Run the 'add' kernel
    add<<<1,N>>>(x, y, z);

    // Copy answer from device to host
    hipMemcpy(c, z, size, hipMemcpyDeviceToHost);

    // Clean device
    hipFree(x);
    hipFree(y);
    hipFree(z);

    // Clean Host
    free(a);
    free(b);
    free(c);


    std::cout << "Answer: " << c[23] << std::endl;

    return 12;
}