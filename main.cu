
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mykernel(void) {

}

__global__ void add(double *a, double *b, double *c){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    c[idx] = a[idx] + b[idx];
}

#define N 2560
#define THREADS_PER_BLOCK 512

int main(void){
    mykernel<<<1,1>>>();
    std::cout << "wow" << std::endl;

    double *a, *b, *c;  // Host variables
    double *x, *y, *z;  // Device variables
    int size = N * sizeof(double);

    hipMalloc((void **)&x, size);
    hipMalloc((void **)&y, size);
    hipMalloc((void **)&z, size);

    // Set Input
    a = (double *)malloc(size);
    b = (double *)malloc(size);
    c = (double *)malloc(size);
    a[23] = 33.249;
    b[23] = -30.741;

    // Copy from host to device
    hipMemcpy(x, a, size, hipMemcpyHostToDevice);
    hipMemcpy(y, b, size, hipMemcpyHostToDevice);

    // Run the 'add' kernel
    add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(x, y, z);

    // Copy answer from device to host
    hipMemcpy(c, z, size, hipMemcpyDeviceToHost);

    // Clean device
    hipFree(x);
    hipFree(y);
    hipFree(z);

    // Clean Host
    free(a);
    free(b);
    free(c);


    std::cout << "Answer: " << c[23] << std::endl;

    return 12;
}